#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define SHARED_MEMORY 1

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernelInclusiveToExclusivePrefixSum(const int n, const int* idata, int* odata)
        {
            int g_index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (g_index >= n)
            {
                return;
            }

            if (g_index > 0)
            {
                odata[g_index] = idata[g_index - 1];
            }
            else if (g_index == 0)
            {
                odata[g_index] = 0;
            }
        }

        __global__ void kernelNaiveInclusivePrefixSumPass(const int n, const int offset, const int* idata, int* odata)
        {
            int g_index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (g_index >= n)
            {
                return;
            }

            if (g_index >= offset)
            {
                odata[g_index] = idata[g_index - offset] + idata[g_index];
            }
            else
            {
                odata[g_index] = idata[g_index];
            }
        }

        __global__ void kernelNaiveInclusivePrefixSumByBlock(const int n, const int* idata, int* odata)
        {
            int g_index = blockIdx.x * blockDim.x + threadIdx.x;

            if (g_index >= n)
            {
                return;
            }

            // allocated on invocation
            extern __shared__ int doubleBuffer[];

            int tx = threadIdx.x;

            int pout = 0, pin = 1;

            // Load input into shared memory.
            doubleBuffer[pout * blockSize + tx] = idata[g_index];
            doubleBuffer[pin * blockSize + tx] = doubleBuffer[pout * blockSize + tx];
            __syncthreads();

            for (int offset = 1; offset < blockSize; offset *= 2)
            {
                // swap double buffer indices
                pout = 1 - pout;
                pin = 1 - pout;

                if (tx >= offset)
                {
                    doubleBuffer[pout * blockSize + tx] = doubleBuffer[pin * blockSize + tx - offset] + doubleBuffer[pin * blockSize + tx];
                }
                else
                {
                    doubleBuffer[pout * blockSize + tx] = doubleBuffer[pin * blockSize + tx];
                }
                __syncthreads();
            }

            // write output
            odata[g_index] = doubleBuffer[pout * blockSize + tx];
        }

        __global__ void kernelNaiveExclusivePrefixSumByBlock(const int n, const int* idata, int* odata)
        {
            int g_index = blockIdx.x * blockDim.x + threadIdx.x;

            if (g_index >= n)
            {
                return;
            }

            // allocated on invocation
            extern __shared__ int doubleBuffer[];

            int tx = threadIdx.x;

            int pout = 0, pin = 1;

            // Load input into shared memory.
            // This is exclusive scan, so shift right by one
            // and set first element to 0
            doubleBuffer[pout * blockSize + tx] = (tx > 0) ? idata[g_index - 1] : 0;
            doubleBuffer[pin * blockSize + tx] = doubleBuffer[pout * blockSize + tx];
            __syncthreads();

            for (int offset = 1; offset < blockSize; offset *= 2)
            {
                // swap double buffer indices
                pout = 1 - pout;
                pin = 1 - pout;

                if (tx >= offset)
                {
                    doubleBuffer[pout * blockSize + tx] = doubleBuffer[pin * blockSize + tx - offset] + doubleBuffer[pin * blockSize + tx];
                }
                else
                {
                    doubleBuffer[pout * blockSize + tx] = doubleBuffer[pin * blockSize + tx];
                }
                __syncthreads();
            }

            // write output
            odata[g_index] = doubleBuffer[pout * blockSize + tx];
        }

        __global__ void kernelExtractBlockSums(const int n, const int numBlocks, const int* idata, int* odata)
        {
            int g_index = blockIdx.x * blockDim.x + threadIdx.x;

            if (g_index >= numBlocks)
            {
                return;
            }

            odata[g_index] = g_index == numBlocks - 1 ? idata[n - 1] : idata[(g_index * blockSize) + blockSize - 1];
        }

        __global__ void kernelAddBlockIncrements(const int n, const int* idataBlockSums, const int* idata, int* odata)
        {
            int g_index = blockIdx.x * blockDim.x + threadIdx.x;

            if (g_index >= n)
            {
                return;
            }

            odata[g_index] = idata[g_index] + idataBlockSums[blockIdx.x];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata)
        {
            #if SHARED_MEMORY
            naiveExclusivePrefixSumSharedMemory(n, idata, odata);
            #else
            naiveExclusivePrefixSum(n, idata, odata);
            #endif
        }

        void naiveExclusivePrefixSum(const int n, const int* idata, int* odata)
        {
            int* dev_bufferA;
            int* dev_bufferB;

            hipMalloc((void**)&dev_bufferA, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_bufferA failed!");

            hipMalloc((void**)&dev_bufferB, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_bufferB failed!");

            hipMemcpy(dev_bufferA, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("memcpy idata to dev_bufferA failed!");

            dim3 blocksPerGrid((n + blockSize - 1) / blockSize);

            timer().startGpuTimer();

            
            for (int offset = 1; offset < n; offset *= 2)
            {
                kernelNaiveInclusivePrefixSumPass<<<blocksPerGrid, blockSize>>>(n, offset, dev_bufferA, dev_bufferB);
                checkCUDAError("kernelNaiveInclusivePrefixSumPass failed!");

                // set the input of the next iteration to the output of this iteration
                std::swap(dev_bufferA, dev_bufferB);
            }

            kernelInclusiveToExclusivePrefixSum<<<blocksPerGrid, blockSize>>>(n, dev_bufferA, dev_bufferB);
            checkCUDAError("kernelInclusiveToExclusivePrefixSum failed!");

            hipMemcpy(odata, dev_bufferB, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("memcpy dev_bufferB to odata failed!");


            timer().endGpuTimer();

            hipFree(dev_bufferA);
            hipFree(dev_bufferB);
        }

        void naiveExclusivePrefixSumSharedMemory(const int n, const int* idata, int* odata)
        {
            int* dev_bufferA;
            int* dev_bufferB;

            hipMalloc((void**)&dev_bufferA, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_bufferA failed!");

            hipMalloc((void**)&dev_bufferB, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_bufferB failed!");

            hipMemcpy(dev_bufferA, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("memcpy idata to dev_bufferA failed!");

            const int sharedMemoryBytes = 2 * blockSize * sizeof(int);

            int numBlocks = (n + blockSize - 1) / blockSize;

            timer().startGpuTimer();


            naiveInclusivePrefixSumAnyNumberOfBlocks(sharedMemoryBytes, n, numBlocks, dev_bufferA, dev_bufferB);

            kernelInclusiveToExclusivePrefixSum<<<(n + blockSize - 1) / blockSize, blockSize>>>(n, dev_bufferB, dev_bufferA);
            checkCUDAError("kernelInclusiveToExclusivePrefixSum failed!");

            hipMemcpy(odata, dev_bufferA, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("memcpy dev_bufferA to odata failed!");


            timer().endGpuTimer();

            hipFree(dev_bufferA);
            hipFree(dev_bufferB);
        }

        // iterative approach is possible if the blockSums buffers are allocated carefully ahead of time, combined with clever indexing of them at each iteration
        // for the sake of submitting this assignement on time, this will have to be explored at a later time
        void naiveInclusivePrefixSumAnyNumberOfBlocks(const int sharedMemoryBytes, const int n, const int numBlocks, int* idata, int* odata)
        {
            kernelNaiveInclusivePrefixSumByBlock<<<numBlocks, blockSize, sharedMemoryBytes>>>(n, idata, odata);
            checkCUDAError("kernelNaiveInclusivePrefixSumByBlock failed!");

            if (numBlocks > 1)
            {
                int* dev_bufferBlockSumsA;
                int* dev_bufferBlockSumsB;

                hipMalloc((void**)&dev_bufferBlockSumsA, sizeof(int) * numBlocks);
                checkCUDAError("hipMalloc dev_bufferBlockSumsA failed!");

                hipMalloc((void**)&dev_bufferBlockSumsB, sizeof(int) * numBlocks);
                checkCUDAError("hipMalloc dev_bufferBlockSumsB failed!");

                int numBlocksForBlockSums = (numBlocks + blockSize - 1) / blockSize;
                kernelExtractBlockSums<<<numBlocksForBlockSums, blockSize>>>(n, numBlocks, odata, dev_bufferBlockSumsA);
                checkCUDAError("kernelExtractBlockSums failed!");

                if (numBlocksForBlockSums > 1)
                {
                    naiveInclusivePrefixSumAnyNumberOfBlocks(sharedMemoryBytes, numBlocks, numBlocksForBlockSums, dev_bufferBlockSumsA, dev_bufferBlockSumsB);

                    kernelInclusiveToExclusivePrefixSum<<<numBlocksForBlockSums, blockSize>>>(numBlocks, dev_bufferBlockSumsB, dev_bufferBlockSumsA);
                    checkCUDAError("kernelInclusiveToExclusivePrefixSum failed!");

                    kernelAddBlockIncrements<<<numBlocks, blockSize>>>(n, dev_bufferBlockSumsA, odata, idata);
                    checkCUDAError("kernelAddBlockIncrements failed!");
                }
                else
                {
                    kernelNaiveExclusivePrefixSumByBlock<<<numBlocksForBlockSums, blockSize, sharedMemoryBytes>>>(numBlocks, dev_bufferBlockSumsA, dev_bufferBlockSumsB);
                    checkCUDAError("kernelNaiveExclusivePrefixSumByBlock failed!");

                    kernelAddBlockIncrements<<<numBlocks, blockSize>>>(n, dev_bufferBlockSumsB, odata, idata);
                    checkCUDAError("kernelAddBlockIncrements failed!");
                }

                hipMemcpy(odata, idata, sizeof(int) * n, hipMemcpyDeviceToDevice);
                checkCUDAError("memcpy idata to odata failed!");

                hipFree(dev_bufferBlockSumsA);
                hipFree(dev_bufferBlockSumsB);
            }
        }
    }
}