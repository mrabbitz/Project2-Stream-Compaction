#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernelInclusiveToExclusivePrefixSum(const int n, const int* idata, int* odata)
        {
            int g_index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (g_index >= n)
            {
                return;
            }

            if (g_index > 0)
            {
                odata[g_index] = idata[g_index - 1];
            }
            else if (g_index == 0)
            {
                odata[g_index] = 0;
            }
        }

        __global__ void kernelNaiveInclusivePrefixSumPass(const int n, const int offset, const int* idata, int* odata)
        {
            int g_index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (g_index >= n)
            {
                return;
            }

            if (g_index >= offset)
            {
                odata[g_index] = idata[g_index - offset] + idata[g_index];
            }
            else
            {
                odata[g_index] = idata[g_index];
            }
        }

        __global__ void kernelNaiveInclusivePrefixSumByBlock(const int n, const int* idata, int* odata)
        {
            // allocated on invocation
            extern __shared__ int doubleBuffer[];

            int g_index = blockIdx.x * blockDim.x + threadIdx.x;
            if (g_index >= n)
            {
                return;
            }

            int tx = threadIdx.x;

            // identify which half of double buffer is read-half and write-half
            int writeBuffer = 0;
            int readBuffer = 1;

            // Load input into shared memory
            // Only need to write to the first half since our first write will be to the second half
            doubleBuffer[tx] = idata[g_index];
            __syncthreads();

            for (int offset = 1; offset < blockSize; offset *= 2)
            {
                // swap double buffer indices
                writeBuffer = 1 - writeBuffer;
                readBuffer = 1 - writeBuffer;

                if (tx >= offset)
                {
                    doubleBuffer[writeBuffer * blockSize + tx] = doubleBuffer[readBuffer * blockSize + tx - offset] + doubleBuffer[readBuffer * blockSize + tx];
                }
                else
                {
                    doubleBuffer[writeBuffer * blockSize + tx] = doubleBuffer[readBuffer * blockSize + tx];
                }
                __syncthreads();
            }

            // write output
            odata[g_index] = doubleBuffer[writeBuffer * blockSize + tx];
        }

        __global__ void kernelNaiveExclusivePrefixSumByBlock(const int n, const int* idata, int* odata)
        {
            // allocated on invocation
            extern __shared__ int doubleBuffer[];

            int g_index = blockIdx.x * blockDim.x + threadIdx.x;
            if (g_index >= n)
            {
                return;
            }

            int tx = threadIdx.x;

            // identify which half of double buffer is read-half and write-half
            int writeBuffer = 0;
            int readBuffer = 1;

            // Load input into shared memory
            // Exclusive scan - shift all elements right by one and set first element to 0
            // Only need to write to the first half since our first write will be to the second half
            doubleBuffer[tx] = (tx > 0) ? idata[g_index - 1] : 0;
            __syncthreads();

            for (int offset = 1; offset < blockSize; offset *= 2)
            {
                // swap double buffer indices
                writeBuffer = 1 - writeBuffer;
                readBuffer = 1 - writeBuffer;

                if (tx >= offset)
                {
                    doubleBuffer[writeBuffer * blockSize + tx] = doubleBuffer[readBuffer * blockSize + tx - offset] + doubleBuffer[readBuffer * blockSize + tx];
                }
                else
                {
                    doubleBuffer[writeBuffer * blockSize + tx] = doubleBuffer[readBuffer * blockSize + tx];
                }
                __syncthreads();
            }

            // write output
            odata[g_index] = doubleBuffer[writeBuffer * blockSize + tx];
        }

        __global__ void kernelExtractBlockSums(const int n, const int numBlocks, const int* idata, int* odata)
        {
            int g_index = blockIdx.x * blockDim.x + threadIdx.x;
            if (g_index >= numBlocks)
            {
                return;
            }

            odata[g_index] = g_index == numBlocks - 1 ? idata[n - 1] : idata[(g_index * blockSize) + blockSize - 1];
        }

        __global__ void kernelAddBlockSumsToBlockData(const int n, const int* idataBlockSums, int* data)
        {
            int g_index = blockIdx.x * blockDim.x + threadIdx.x;
            if (g_index >= n)
            {
                return;
            }

            data[g_index] += idataBlockSums[blockIdx.x];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata, bool useSharedMemory)
        {
            if (useSharedMemory)
            {
                naiveExclusivePrefixSumSharedMemory(n, idata, odata);
            }
            else
            {
                naiveExclusivePrefixSum(n, idata, odata);
            }
        }

        void naiveExclusivePrefixSum(const int n, const int* idata, int* odata)
        {
            int* dev_bufferA;
            int* dev_bufferB;

            hipMalloc((void**)&dev_bufferA, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_bufferA failed!");

            hipMalloc((void**)&dev_bufferB, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_bufferB failed!");

            hipMemcpy(dev_bufferA, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("memcpy idata to dev_bufferA failed!");

            int blocksPerGrid = (n + blockSize - 1) / blockSize;

            timer().startGpuTimer();

            
            for (int offset = 1; offset < n; offset *= 2)
            {
                kernelNaiveInclusivePrefixSumPass<<<blocksPerGrid, blockSize>>>(n, offset, dev_bufferA, dev_bufferB);
                checkCUDAError("kernelNaiveInclusivePrefixSumPass failed!");

                // set the input of the next iteration to the output of this iteration
                std::swap(dev_bufferA, dev_bufferB);
            }

            kernelInclusiveToExclusivePrefixSum<<<blocksPerGrid, blockSize>>>(n, dev_bufferA, dev_bufferB);
            checkCUDAError("kernelInclusiveToExclusivePrefixSum failed!");


            timer().endGpuTimer();

            hipMemcpy(odata, dev_bufferB, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("memcpy dev_bufferB to odata failed!");

            hipFree(dev_bufferA);
            hipFree(dev_bufferB);
            checkCUDAError("hipFree failed!");
        }

        void naiveExclusivePrefixSumSharedMemory(const int n, const int* idata, int* odata)
        {
            int* dev_bufferA;
            int* dev_bufferB;

            hipMalloc((void**)&dev_bufferA, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_bufferA failed!");

            hipMalloc((void**)&dev_bufferB, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_bufferB failed!");

            hipMemcpy(dev_bufferA, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("memcpy idata to dev_bufferA failed!");

            int blocksPerGrid = (n + blockSize - 1) / blockSize;

            int sharedMemoryBytes = 2 * blockSize * sizeof(int);

            timer().startGpuTimer();


            naiveInclusivePrefixSumAnyNumberOfBlocks(sharedMemoryBytes, n, blocksPerGrid, dev_bufferA, dev_bufferB);

            kernelInclusiveToExclusivePrefixSum<<<blocksPerGrid, blockSize>>>(n, dev_bufferB, dev_bufferA);
            checkCUDAError("kernelInclusiveToExclusivePrefixSum failed!");


            timer().endGpuTimer();

            hipMemcpy(odata, dev_bufferA, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("memcpy dev_bufferA to odata failed!");

            hipFree(dev_bufferA);
            hipFree(dev_bufferB);
            checkCUDAError("hipFree failed!");
        }

        // iterative approach is possible if the blockSums buffers are allocated carefully ahead of time, combined with clever indexing of them at each iteration
        // for the sake of submitting this assignement on time, this will have to be explored at a later time
        void naiveInclusivePrefixSumAnyNumberOfBlocks(const int sharedMemoryBytes, const int n, const int numBlocks, int* idata, int* odata)
        {
            kernelNaiveInclusivePrefixSumByBlock<<<numBlocks, blockSize, sharedMemoryBytes>>>(n, idata, odata);
            checkCUDAError("kernelNaiveInclusivePrefixSumByBlock failed!");

            if (numBlocks > 1)
            {
                int* dev_bufferBlockSumsA;
                int* dev_bufferBlockSumsB;

                hipMalloc((void**)&dev_bufferBlockSumsA, sizeof(int) * numBlocks);
                checkCUDAError("hipMalloc dev_bufferBlockSumsA failed!");

                hipMalloc((void**)&dev_bufferBlockSumsB, sizeof(int) * numBlocks);
                checkCUDAError("hipMalloc dev_bufferBlockSumsB failed!");

                int numBlocksForBlockSums = (numBlocks + blockSize - 1) / blockSize;

                kernelExtractBlockSums<<<numBlocksForBlockSums, blockSize>>>(n, numBlocks, odata, dev_bufferBlockSumsA);
                checkCUDAError("kernelExtractBlockSums failed!");

                if (numBlocksForBlockSums > 1)
                {
                    naiveInclusivePrefixSumAnyNumberOfBlocks(sharedMemoryBytes, numBlocks, numBlocksForBlockSums, dev_bufferBlockSumsA, dev_bufferBlockSumsB);

                    kernelInclusiveToExclusivePrefixSum<<<numBlocksForBlockSums, blockSize>>>(numBlocks, dev_bufferBlockSumsB, dev_bufferBlockSumsA);
                    checkCUDAError("kernelInclusiveToExclusivePrefixSum failed!");

                    kernelAddBlockSumsToBlockData<<<numBlocks, blockSize>>>(n, dev_bufferBlockSumsA, odata);
                    checkCUDAError("kernelAddBlockSumsToBlockData failed!");
                }
                else
                {
                    kernelNaiveExclusivePrefixSumByBlock<<<numBlocksForBlockSums, blockSize, sharedMemoryBytes>>>(numBlocks, dev_bufferBlockSumsA, dev_bufferBlockSumsB);
                    checkCUDAError("kernelNaiveExclusivePrefixSumByBlock failed!");

                    kernelAddBlockSumsToBlockData<<<numBlocks, blockSize>>>(n, dev_bufferBlockSumsB, odata);
                    checkCUDAError("kernelAddBlockSumsToBlockData failed!");
                }

                hipFree(dev_bufferBlockSumsA);
                hipFree(dev_bufferBlockSumsB);
                checkCUDAError("hipFree failed!");
            }
        }
    }
}