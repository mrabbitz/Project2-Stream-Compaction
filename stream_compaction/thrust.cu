#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {

            int* dev_idata;
            int* dev_odata;

            hipMalloc((void**)&dev_idata, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_idata failed!");

            hipMalloc((void**)&dev_odata, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_odata failed!");

            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("memcpy idata to dev_idata failed!");

            thrust::device_ptr<int> dev_thrust_idata(dev_idata);
            thrust::device_ptr<int> dev_thrust_odata(dev_odata);

            timer().startGpuTimer();


            thrust::exclusive_scan(dev_thrust_idata, dev_thrust_idata + n, dev_thrust_odata);


            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("memcpy dev_odata to odata failed!");

            hipFree(dev_idata);
            hipFree(dev_odata);
            checkCUDAError("hipFree failed!");
        }
    }
}
